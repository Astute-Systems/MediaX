#include "hip/hip_runtime.h"
//
// Copyright (c) 2023, DefenceX PTY LTD
//
// This file is part of the VivoeX project developed by DefenceX.
//
// Licensed under the Attribution-NonCommercial 4.0 International (CC BY-NC 4.0)
// License. See the LICENSE file in the project root for full license details.
//
/// \brief Functions to convert between different colour spaces, these functions are optimized for Nvidia CUDA when
/// available.
///
/// \file colourspace_cuda.cc
///

#include <hip/hip_runtime.h>
#include <stdint.h>

#include <iostream>

#include "utils/colourspace_cuda.h"

namespace mediax::video {

ColourSpaceCuda::ColourSpaceCuda() {
  // Initialise CUDA
  hipError_t cudaStatus = hipSetDevice(0);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
  }
}

__global__ void RgbToYuvKernel<<<dimGrid, dimBlock>>>(height, width, rgb, yuv) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) return;

  int rgbIndex = y * width * 3 + x * 3;
  int yuvIndex = y * width * 2 + x * 2;

  int r = rgb[rgbIndex];
  int g = rgb[rgbIndex + 1];
  int b = rgb[rgbIndex + 2];

  int yVal = (int)(0.299f * r + 0.587f * g + 0.114f * b);
  int uVal = (int)(-0.14713f * r - 0.28886f * g + 0.436f * b);
  int vVal = (int)(0.615f * r - 0.51499f * g - 0.10001f * b);

  yVal = min(max(0, yVal), 255);
  uVal = min(max(0, uVal + 128), 255);
  vVal = min(max(0, vVal + 128), 255);

  yuv[yuvIndex] = (uint8_t)yVal;
  yuv[yuvIndex | 1] = (uint8_t)uVal;
  yuv[yuvIndex & ~1] = (uint8_t)vVal;
}

int RgbToYuv(uint32_t height, uint32_t width, uint8_t *rgb, uint8_t *yuv) const {
  int block_size = 32;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
  RgbToYuvKernel<<<dimGrid, dimBlock>>>(height, width, rgb, yuv);
  hipDeviceSynchronize();
}

__global__ void RgbToMono8Kernel<<<gridDim, blockDim>>>(height, width, dev_rgb, dev_mono8) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) return;

  int rgbIndex = y * width * 3 + x * 3;
  int mono8Index = y * width + x;

  int r = dev_rgb[rgbIndex];
  int g = dev_rgb[rgbIndex + 1];
  int b = dev_rgb[rgbIndex + 2];

  int mono8Val = (int)(0.299f * r + 0.587f * g + 0.114f * b);

  dev_mono8[mono8Index] = (uint8_t)mono8Val;
}

int RgbToMono8(uint32_t height, uint32_t width, uint8_t *rgb, uint8_t *mono8) const {
  if (!rgb || !mono8) {
    return;
  }

  uint8_t *dev_rgb, *dev_mono8;
  size_t rgb_size = height * width * 3 * sizeof(uint8_t);
  size_t mono8_size = height * width * sizeof(uint8_t);

  // Allocate memory on GPU
  hipMalloc((void **)&dev_rgb, rgb_size);
  hipMalloc((void **)&dev_mono8, mono8_size);

  // Copy input data to GPU
  hipMemcpy(dev_rgb, rgb, rgb_size, hipMemcpyHostToDevice);

  // Launch CUDA kernel
  dim3 blockDim(16, 16);
  dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
  RgbToMono8Kernel<<<gridDim, blockDim>>>(height, width, dev_rgb, dev_mono8);

  // Copy result back to host
  hipMemcpy(mono8, dev_mono8, mono8_size, hipMemcpyDeviceToHost);

  // Clean up
  hipFree(dev_rgb);
  hipFree(dev_mono8);
}

__global__ void YuvToRgbKernel(uint32_t height, uint32_t width, uint8_t *yuv, uint8_t *rgb) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) return;

  int yuvIndex = y * width * 2 + x * 2;
  int rgbIndex = y * width * 3 + x * 3;

  int yVal = yuv[yuvIndex];
  int uVal = yuv[yuvIndex | 1] - 128;
  int vVal = yuv[yuvIndex & ~1] - 128;

  int c = yVal - 16;
  int d = uVal;
  int e = vVal;

  int r = (298 * c + 409 * e + 128) >> 8;
  int g = (298 * c - 100 * d - 208 * e + 128) >> 8;
  int b = (298 * c + 516 * d + 128) >> 8;

  rgb[rgbIndex] = min(max(r, 0), 255);
  rgb[rgbIndex + 1] = min(max(g, 0), 255);
  rgb[rgbIndex + 2] = min(max(b, 0), 255);
}

int ColourSpaceCuda::YuvToRgb(uint32_t height, uint32_t width, uint8_t *yuv, uint8_t *rgb) const {
  if (!rgb || !yuv) {
    return;
  }

  uint8_t *dev_yuv, *dev_rgb;
  size_t yuv_size = height * width * 2 * sizeof(uint8_t);
  size_t rgb_size = height * width * 3 * sizeof(uint8_t);

  // Allocate memory on GPU
  hipMalloc((void **)&dev_yuv, yuv_size);
  hipMalloc((void **)&dev_rgb, rgb_size);

  // Copy input data to GPU
  hipMemcpy(dev_yuv, yuv, yuv_size, hipMemcpyHostToDevice);

  // Launch CUDA kernel
  dim3 blockDim(16, 16);
  dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
  YuvToRgbKernel<<<gridDim, blockDim>>>(height, width, dev_yuv, dev_rgb);

  // Copy result back to host
  hipMemcpy(rgb, dev_rgb, rgb_size, hipMemcpyDeviceToHost);

  // Clean up
  hipFree(dev_yuv);
  hipFree(dev_rgb);
}

__global__ void yuvToRgbaKernel(uint32_t height, uint32_t width, uint8_t *yuv, uint8_t *rgba) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height) {
    int yIndex = y * width + x;
    int uIndex = (y / 2) * (width / 2) + (x / 2) + (height * width);
    int vIndex = (y / 2) * (width / 2) + (x / 2) + (height * width) + ((height * width) / 4);
    int r, g, b;
    int yValue = yuv[yIndex];
    int uValue = yuv[uIndex];
    int vValue = yuv[vIndex];
    uValue -= 128;
    vValue -= 128;
    r = (int)(yValue + 1.13983f * vValue);
    g = (int)(yValue - 0.39465f * uValue - 0.58060f * vValue);
    b = (int)(yValue + 2.03211f * uValue);
    r = min(max(0, r), 255);
    g = min(max(0, g), 255);
    b = min(max(0, b), 255);
    int rgbaIndex = yIndex * 4;
    rgba[rgbaIndex] = (uint8_t)r;
    rgba[rgbaIndex + 1] = (uint8_t)g;
    rgba[rgbaIndex + 2] = (uint8_t)b;
    rgba[rgbaIndex + 3] = 255;
  }
}

int ColourSpaceCuda::YuvToRgba(uint32_t height, uint32_t width, uint8_t *yuv, uint8_t *rgba) const {
  int block_size = 32;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
  yuvToRgbaKernel<<<dimGrid, dimBlock>>>(height, width, yuv, rgba);
  hipDeviceSynchronize();
}

__global__ void rgbToYuvKernel(uint32_t height, uint32_t width, uint8_t *rgb, uint8_t *yuv) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height) {
    int index = y * width + x;
    int r = rgb[index * 3];
    int g = rgb[index * 3 + 1];
    int b = rgb[index * 3 + 2];
    int yValue = (int)(0.299f * r + 0.587f * g + 0.114f * b);
    int uValue = (int)(-0.14713f * r - 0.28886f * g + 0.436f * b);
    int vValue = (int)(0.615f * r - 0.51499f * g - 0.10001f * b);
    yValue = min(max(0, yValue), 255);
    uValue = min(max(0, uValue + 128), 255);
    vValue = min(max(0, vValue + 128), 255);
    int yIndex = y * width + x;
    int uIndex = (y / 2) * (width / 2) + (x / 2) + (height * width);
    int vIndex = (y / 2) * (width / 2) + (x / 2) + (height * width) + ((height * width) / 4);
    yuv[yIndex] = (uint8_t)yValue;
    yuv[uIndex] = (uint8_t)uValue;
    yuv[vIndex] = (uint8_t)vValue;
  }
}

int RgbToYuv(uint32_t height, uint32_t width, uint8_t *rgb, uint8_t *yuv) const {
  int block_size = 32;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
  rgbToYuvKernel<<<dimGrid, dimBlock>>>(height, width, rgb, yuv);
  hipDeviceSynchronize();
}

__global__ void rgbaToYuvKernel(uint32_t height, uint32_t width, uint8_t *rgba, uint8_t *yuv) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height) {
    int index = y * width + x;
    int r = rgba[index * 4];
    int g = rgba[index * 4 + 1];
    int b = rgba[index * 4 + 2];
    int yValue = (int)(0.299f * r + 0.587f * g + 0.114f * b);
    int uValue = (int)(-0.14713f * r - 0.28886f * g + 0.436f * b);
    int vValue = (int)(0.615f * r - 0.51499f * g - 0.10001f * b);
    yValue = min(max(0, yValue), 255);
    uValue = min(max(0, uValue + 128), 255);
    vValue = min(max(0, vValue + 128), 255);
    int yIndex = y * width + x;
    int uIndex = (y / 2) * (width / 2) + (x / 2) + (height * width);
    int vIndex = (y / 2) * (width / 2) + (x / 2) + (height * width) + ((height * width) / 4);
    yuv[yIndex] = (uint8_t)yValue;
    yuv[uIndex] = (uint8_t)uValue;
    yuv[vIndex] = (uint8_t)vValue;
  }
}

int ColourSpaceCuda::RgbaToYuv(uint32_t height, uint32_t width, uint8_t *rgba, uint8_t *yuv) {
  int block_size = 32;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
  rgbaToYuvKernel<<<dimGrid, dimBlock>>>(height, width, rgba, yuv);
  hipDeviceSynchronize();
  return 0;
}

}  // namespace mediax::video
