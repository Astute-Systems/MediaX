#include "hip/hip_runtime.h"
//
// Copyright (c) 2023, DefenceX PTY LTD
//
// This file is part of the VivoeX project developed by DefenceX.
//
// Licensed under the Attribution-NonCommercial 4.0 International (CC BY-NC 4.0)
// License. See the LICENSE file in the project root for full license details.
//
/// \brief Functions to convert between different colour spaces, these functions are optimized for Nvidia CUDA when
/// available.
///
/// \file colourspace_cuda.cc
///

#include <hip/hip_runtime.h>
#include <stdint.h>

#include <iostream>

#include "utils/colourspace_cpu.h"

namespace video {

// CUDA kernel to convert YUV to RGB
__global__ void YuvToRgbKernel(uint32_t height, uint32_t width, uint8_t *yuv, uint8_t *rgb) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) return;

  int yuvIndex = y * width * 2 + x * 2;
  int rgbIndex = y * width * 3 + x * 3;

  int yVal = yuv[yuvIndex];
  int uVal = yuv[yuvIndex | 1] - 128;
  int vVal = yuv[yuvIndex & ~1] - 128;

  int c = yVal - 16;
  int d = uVal;
  int e = vVal;

  int r = (298 * c + 409 * e + 128) >> 8;
  int g = (298 * c - 100 * d - 208 * e + 128) >> 8;
  int b = (298 * c + 516 * d + 128) >> 8;

  rgb[rgbIndex] = min(max(r, 0), 255);
  rgb[rgbIndex + 1] = min(max(g, 0), 255);
  rgb[rgbIndex + 2] = min(max(b, 0), 255);
}

void ColourSpaceCuda::YuvToRgb(uint32_t height, uint32_t width, uint8_t *yuv, uint8_t *rgb) const {
  if (!rgb || !yuv) {
    return;
  }

  uint8_t *dev_yuv, *dev_rgb;
  size_t yuv_size = height * width * 2 * sizeof(uint8_t);
  size_t rgb_size = height * width * 3 * sizeof(uint8_t);

  // Allocate memory on GPU
  hipMalloc((void **)&dev_yuv, yuv_size);
  hipMalloc((void **)&dev_rgb, rgb_size);

  // Copy input data to GPU
  hipMemcpy(dev_yuv, yuv, yuv_size, hipMemcpyHostToDevice);

  // Launch CUDA kernel
  dim3 blockDim(16, 16);
  dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
  YuvToRgbKernel<<<gridDim, blockDim>>>(height, width, dev_yuv, dev_rgb);

  // Copy result back to host
  hipMemcpy(rgb, dev_rgb, rgb_size, hipMemcpyDeviceToHost);

  // Clean up
  hipFree(dev_yuv);
  hipFree(dev_rgb);
}

__global__ void yuvToRgbaKernel(uint32_t height, uint32_t width, uint8_t *yuv, uint8_t *rgba) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height) {
    int yIndex = y * width + x;
    int uIndex = (y / 2) * (width / 2) + (x / 2) + (height * width);
    int vIndex = (y / 2) * (width / 2) + (x / 2) + (height * width) + ((height * width) / 4);
    int r, g, b;
    int yValue = yuv[yIndex];
    int uValue = yuv[uIndex];
    int vValue = yuv[vIndex];
    uValue -= 128;
    vValue -= 128;
    r = (int)(yValue + 1.13983f * vValue);
    g = (int)(yValue - 0.39465f * uValue - 0.58060f * vValue);
    b = (int)(yValue + 2.03211f * uValue);
    r = min(max(0, r), 255);
    g = min(max(0, g), 255);
    b = min(max(0, b), 255);
    int rgbaIndex = yIndex * 4;
    rgba[rgbaIndex] = (uint8_t)r;
    rgba[rgbaIndex + 1] = (uint8_t)g;
    rgba[rgbaIndex + 2] = (uint8_t)b;
    rgba[rgbaIndex + 3] = 255;
  }
}

void ColourSpaceCuda::YuvToRgba(uint32_t height, uint32_t width, uint8_t *yuv, uint8_t *rgba) const {
  int block_size = 32;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
  yuvToRgbaKernel<<<dimGrid, dimBlock>>>(height, width, yuv, rgba);
  hipDeviceSynchronize();
}

__global__ void rgbToYuvKernel(uint32_t height, uint32_t width, uint8_t *rgb, uint8_t *yuv) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height) {
    int index = y * width + x;
    int r = rgb[index * 3];
    int g = rgb[index * 3 + 1];
    int b = rgb[index * 3 + 2];
    int yValue = (int)(0.299f * r + 0.587f * g + 0.114f * b);
    int uValue = (int)(-0.14713f * r - 0.28886f * g + 0.436f * b);
    int vValue = (int)(0.615f * r - 0.51499f * g - 0.10001f * b);
    yValue = min(max(0, yValue), 255);
    uValue = min(max(0, uValue + 128), 255);
    vValue = min(max(0, vValue + 128), 255);
    int yIndex = y * width + x;
    int uIndex = (y / 2) * (width / 2) + (x / 2) + (height * width);
    int vIndex = (y / 2) * (width / 2) + (x / 2) + (height * width) + ((height * width) / 4);
    yuv[yIndex] = (uint8_t)yValue;
    yuv[uIndex] = (uint8_t)uValue;
    yuv[vIndex] = (uint8_t)vValue;
  }
}

void RgbToYuv(uint32_t height, uint32_t width, uint8_t *rgb, uint8_t *yuv) const {
  int block_size = 32;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
  rgbToYuvKernel<<<dimGrid, dimBlock>>>(height, width, rgb, yuv);
  hipDeviceSynchronize();
}

__global__ void rgbaToYuvKernel(uint32_t height, uint32_t width, uint8_t *rgba, uint8_t *yuv) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x < width && y < height) {
    int index = y * width + x;
    int r = rgba[index * 4];
    int g = rgba[index * 4 + 1];
    int b = rgba[index * 4 + 2];
    int yValue = (int)(0.299f * r + 0.587f * g + 0.114f * b);
    int uValue = (int)(-0.14713f * r - 0.28886f * g + 0.436f * b);
    int vValue = (int)(0.615f * r - 0.51499f * g - 0.10001f * b);
    yValue = min(max(0, yValue), 255);
    uValue = min(max(0, uValue + 128), 255);
    vValue = min(max(0, vValue + 128), 255);
    int yIndex = y * width + x;
    int uIndex = (y / 2) * (width / 2) + (x / 2) + (height * width);
    int vIndex = (y / 2) * (width / 2) + (x / 2) + (height * width) + ((height * width) / 4);
    yuv[yIndex] = (uint8_t)yValue;
    yuv[uIndex] = (uint8_t)uValue;
    yuv[vIndex] = (uint8_t)vValue;
  }
}

void ColourSpaceCuda::RgbaToYuv(uint32_t height, uint32_t width, uint8_t *rgba, uint8_t *yuv) {
  int block_size = 32;
  dim3 dimBlock(block_size, block_size);
  dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
  rgbaToYuvKernel<<<dimGrid, dimBlock>>>(height, width, rgba, yuv);
  hipDeviceSynchronize();
}

}  // namespace video
