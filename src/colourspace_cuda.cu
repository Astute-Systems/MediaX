#include "hip/hip_runtime.h"

//
// Copyright (C) 2023 DefenceX PTY LTD. All rights reserved.
//
// This software is distributed under the included copyright license.
// Any redistribution or reproduction, in part or in whole, in any form or medium, is strictly prohibited without the
// prior written consent of DefenceX PTY LTD.
//
// For any inquiries or concerns, please contact:
// DefenceX PTY LTD
// Email: enquiries@defencex.ai
//
/// \brief Functions to convert between different colour spaces, these functions are optimized for Nvidia CUDA when
/// available.
///
/// \file colourspace_cuda.cc
///

#include <hip/hip_runtime.h>
#include <stdint.h>

#include <iostream>

namespace video::cuda {

// CUDA kernel to convert YUV to RGB
__global__ void YuvToRgbKernel(uint32_t height, uint32_t width, uint8_t *yuv, uint8_t *rgb) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) return;

  int yuvIndex = y * width * 2 + x * 2;
  int rgbIndex = y * width * 3 + x * 3;

  int yVal = yuv[yuvIndex];
  int uVal = yuv[yuvIndex | 1] - 128;
  int vVal = yuv[yuvIndex & ~1] - 128;

  int c = yVal - 16;
  int d = uVal;
  int e = vVal;

  int r = (298 * c + 409 * e + 128) >> 8;
  int g = (298 * c - 100 * d - 208 * e + 128) >> 8;
  int b = (298 * c + 516 * d + 128) >> 8;

  rgb[rgbIndex] = min(max(r, 0), 255);
  rgb[rgbIndex + 1] = min(max(g, 0), 255);
  rgb[rgbIndex + 2] = min(max(b, 0), 255);
}

void YuvToRgb(uint32_t height, uint32_t width, uint8_t *yuv, uint8_t *rgb) {
  if (!rgb || !yuv) {
    return;
  }

  uint8_t *dev_yuv, *dev_rgb;
  size_t yuv_size = height * width * 2 * sizeof(uint8_t);
  size_t rgb_size = height * width * 3 * sizeof(uint8_t);

  // Allocate memory on GPU
  hipMalloc((void **)&dev_yuv, yuv_size);
  hipMalloc((void **)&dev_rgb, rgb_size);

  // Copy input data to GPU
  hipMemcpy(dev_yuv, yuv, yuv_size, hipMemcpyHostToDevice);

  // Launch CUDA kernel
  dim3 blockDim(16, 16);
  dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
  YuvToRgbKernel<<<gridDim, blockDim>>>(height, width, dev_yuv, dev_rgb);

  // Copy result back to host
  hipMemcpy(rgb, dev_rgb, rgb_size, hipMemcpyDeviceToHost);

  // Clean up
  hipFree(dev_yuv);
  hipFree(dev_rgb);
}

__global__ void YuvToRgbKernel(uint32_t height, uint32_t width, uint8_t *yuv, uint8_t *rgb) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    int index = y * width + x;
    int yIndex = index;
    int uIndex = (index / 4) + (height * width);
    int vIndex = (index / 4) + (height * width) + ((height * width) / 4);

    int Y = yuv[yIndex];
    int U = yuv[uIndex];
    int V = yuv[vIndex];

    int R = (int)(Y + 1.370705 * (V - 128));
    int G = (int)(Y - 0.698001 * (V - 128) - 0.337633 * (U - 128));
    int B = (int)(Y + 1.732446 * (U - 128));

    R = (R < 0) ? 0 : ((R > 255) ? 255 : R);
    G = (G < 0) ? 0 : ((G > 255) ? 255 : G);
    B = (B < 0) ? 0 : ((B > 255) ? 255 : B);

    int rgbIndex = index * 3;
    rgb[rgbIndex] = (uint8_t)R;
    rgb[rgbIndex + 1] = (uint8_t)G;
    rgb[rgbIndex + 2] = (uint8_t)B;
  }
}

void YuvToRgb(uint32_t height, uint32_t width, uint8_t *yuv, uint8_t *rgb) {
  uint8_t *d_yuv, *d_rgb;
  size_t size = height * width * 3 * sizeof(uint8_t);
  hipMalloc((void **)&d_yuv, size);
  hipMalloc((void **)&d_rgb, size);

  hipMemcpy(d_yuv, yuv, size, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

  YuvToRgbKernel<<<numBlocks, threadsPerBlock>>>(height, width, d_yuv, d_rgb);

  hipMemcpy(rgb, d_rgb, size, hipMemcpyDeviceToHost);

  hipFree(d_yuv);
  hipFree(d_rgb);
}

__global__ void RgbToYuvKernel(uint32_t height, uint32_t width, uint8_t *rgb, uint8_t *yuv) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x < width && y < height) {
    int index = y * width + x;
    int rgbIndex = index * 3;
    int yIndex = index;
    int uIndex = (index / 4) + (height * width);
    int vIndex = (index / 4) + (height * width) + ((height * width) / 4);

    int R = rgb[rgbIndex];
    int G = rgb[rgbIndex + 1];
    int B = rgb[rgbIndex + 2];

    int Y = (int)(0.299 * R + 0.587 * G + 0.114 * B);
    int U = (int)((B - Y) * 0.565 + 128);
    int V = (int)((R - Y) * 0.713 + 128);

    Y = (Y < 0) ? 0 : ((Y > 255) ? 255 : Y);
    U = (U < 0) ? 0 : ((U > 255) ? 255 : U);
    V = (V < 0) ? 0 : ((V > 255) ? 255 : V);

    yuv[yIndex] = (uint8_t)Y;
    yuv[uIndex] = (uint8_t)U;
    yuv[vIndex] = (uint8_t)V;
  }
}

void RgbToYuv(uint32_t height, uint32_t width, uint8_t *rgb, uint8_t *yuv) {
  uint8_t *d_rgb, *d_yuv;
  size_t size = height * width * 3 * sizeof(uint8_t);
  hipMalloc((void **)&d_rgb, size);
  hipMalloc((void **)&d_yuv, size);

  hipMemcpy(d_rgb, rgb, size, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (height + threadsPerBlock.y - 1) / threadsPerBlock.y);

  RgbToYuvKernel<<<numBlocks, threadsPerBlock>>>(height, width, d_rgb, d_yuv);

  hipMemcpy(yuv, d_yuv, size, hipMemcpyDeviceToHost);

  hipFree(d_rgb);
  hipFree(d_yuv);
}

}  // namespace video::cuda
